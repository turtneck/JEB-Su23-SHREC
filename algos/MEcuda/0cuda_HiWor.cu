
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__
void hello( void )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Which one am I?
    printf( "Hello world from thread %d!\n", i ); // What do I do?
}

int main( void ) {
    printf( "Running Kernel A\n" );
    hello<<<1,1>>>( );
    hipDeviceSynchronize( );

    printf( "\n\nRunning Kernel B\n" );
    hello<<<1,32>>>( );
    hipDeviceSynchronize( );

    printf( "\n\nRunning Kernel C\n" );
    hello<<<8,32>>>( );
    hipDeviceSynchronize( );

    return 0;
}