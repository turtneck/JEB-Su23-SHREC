#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <sys\timeb.h>
#include "main.h"
#include "lodepng.h"
#include <string.h>
// For an M (rows) by N (columns) maze
# define M 1001
# define N 1001

void postProcessing(char* inputImage);

__shared__ Point* new_points;

__device__ bool insertPoint(Point array[2 * (N < M ? N : M)], Point point);

__global__ void checkPoint(Point* d_points, Point* d_new_points, unsigned char* image, unsigned nb_threads, int* insert_index, int run_num) //bool* maze
{
	for (int index = (blockIdx.x * nb_threads + threadIdx.x); index < N; index += nb_threads) {
		int pos[4] = { -1 };

		// If Point is empty (ie, still at initial Point(-1,-1) value), skip process but still copy back results
		if (d_points[index].getR() >= 0) {
			// Get row and col for current point
			int row = d_points[index].getR();
			int col = d_points[index].getC();

			// If point above is in bounds and not a wall
			if (row - 1 >= 0 && image[(row - 1) * 4 * N + col * 4 + 2] > 250) {
				// Insert in shared array and get insertion index
				if (insertPoint(new_points, Point((row - 1), col))) {
					pos[0] = atomicAdd(&insert_index[0], 1);
					new_points[pos[0]] = Point((row - 1), col);
					image[(row - 1) * 4 * N + col * 4] = 255 - run_num;
					image[(row - 1) * 4 * N + col * 4 + 1] = 0;
					image[(row - 1) * 4 * N + col * 4 + 2] = 0;
				}
			}

			
			// If point to the left is in bounds and not a wall
			if (col - 1 >= 0 && image[row * 4 * N + (col - 1) * 4 + 2] > 250) {
				// Insert in shared array and get insertion index
				if (insertPoint(new_points, Point(row, (col - 1)))) {
					pos[1] = atomicAdd(&insert_index[0], 1);
					new_points[pos[1]] = Point(row, (col - 1));
					image[row * 4 * N + (col - 1) * 4] = 255 - run_num;
					image[row * 4 * N + (col - 1) * 4 + 1] = 0;
					image[row * 4 * N + (col - 1) * 4 + 2] = 0;
				}
			}

			// If point below is in bounds and not a wall
			if (row + 1 < M  && image[(row + 1) * 4 * N + col * 4 + 2] > 250) {
				// Insert in shared array and get insertion index
				if (insertPoint(new_points, Point((row + 1), col))) {
					pos[2] = atomicAdd(&insert_index[0], 1);
					new_points[pos[2]] = Point((row + 1), col);
					image[(row + 1) * 4 * N + col * 4] = 255 - run_num;
					image[(row + 1) * 4 * N + col * 4 + 1] = 0;
					image[(row + 1) * 4 * N + col * 4 + 2] = 0;
				}
			}

			// If point to the right is in bounds and not a wall
			if (col + 1 < N && image[row * 4 * N + (col + 1) * 4 + 2] > 250) {
				// Insert in shared array and get insertion index
				if (insertPoint(new_points, Point(row, (col + 1)))) {
					pos[3] = atomicAdd(&insert_index[0], 1);
					new_points[pos[3]] = Point(row, (col + 1));
					image[row * 4 * N + (col + 1) * 4] = 255 - run_num;
					image[row * 4 * N + (col + 1) * 4 + 1] = 0;
					image[row * 4 * N + (col + 1) * 4 + 2] = 0;
				}
			}
		}

		for (int k = 0; k < 4; k++) {
			if (pos[k] != -1) {
				d_new_points[pos[k]] = new_points[pos[k]];
			}
		}
	}
}

__global__ void shared_initialize() {
	__shared__ Point i[2 * (N < M ? N : M)];

	for (int j = 0; j < 2 * (N < M ? N : M); j++) {
		i[j] = Point(-1, -1);
	}

	new_points = (Point*)&i;
}

// Insert given point at the first available position in the given array (avoiding duplicate points)
__device__ bool insertPoint(Point array[2 * (N < M ? N : M)], Point point) {
	int i;
	// Cycle through array points until the end or an empty point (ie, still at initial Point(-1,-1) value) is reached
	for (i = 0; i < 2 * (N < M ? N : M) && array[i].getR() >= 0; i++) {
		// If duplicate point found (ie, point we want to insert is already in the array) do nothing and return
		if (point.getR() == array[i].getR() && point.getC() == array[i].getC()) {
			return false;
		}
	}
	return true;
}

int main(int argc, char* argv[])
{
	struct timeb start_time, end_time, cuda_start, cuda_end;
	double cuda_total = 0, total_time = 0;
	ftime(&start_time);
	const int diagonalSize = 2 * (N < M ? N : M);

	if (argc < 2) {
		printf("Invalid arguments! Usage: ./ParallelMazeSolver <name of input png> (optional)<number of threads>\n");
		return -1;
	}

	char* input_filename = argv[1];
	unsigned total_threads = diagonalSize;
	if (argc == 3) total_threads = atoi(argv[2]);
	unsigned nb_threads = total_threads;
	unsigned nb_blocks = 1;
	bool pathFound = false;

	// Max threads per block is 1024
	while (nb_threads > 1024) {
		nb_blocks++;
		nb_threads = total_threads / nb_blocks;
	}

	char* output_filename = (char*)malloc(strlen(input_filename));
	for (int i = 0; i < strlen(input_filename) - 7; i++) {
		output_filename[i] = input_filename[i];
	}
	output_filename[strlen(input_filename) - 7] = 'p';
	output_filename[strlen(input_filename) - 6] = '.';
	output_filename[strlen(input_filename) - 5] = 'p';
	output_filename[strlen(input_filename) - 4] = 'n';
	output_filename[strlen(input_filename) - 3] = 'g';
	output_filename[strlen(input_filename) - 2] = '\0';

	unsigned error;
	unsigned char* image = (unsigned char*)malloc(N * M * sizeof(unsigned char) * 4);
	unsigned char* image_copy;
	unsigned image_width, image_height;

	// Decode image
	error = lodepng_decode32_file(&image, &image_width, &image_height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error)); 

	printf("Input file: %s, maze width: %d, maze height: %d\n", input_filename, image_width, image_height);
	printf("Number of blocks: %d, number of threads: %d\n", nb_blocks, nb_threads);

	// Check that maze is non-empty
	if (image_width * image_height != 0) {
		// Array of points to be visited at each iteration, initialized with all Point(-1,-1) entries
		Point points[diagonalSize];
		// Set first point to be visited - the arrival point (because we are backtracking), the last point of the maze (assuming square maze)
		points[0] = Point(M - 1, N - 1);

		hipMallocManaged((void**)& image_copy, image_width * image_height * 4 * sizeof(unsigned char));
		hipMemcpy(image_copy, image, image_width * image_height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);

		// Cuda copies of the points to be visited and resulting new points to visit for next iteration
		Point* d_points, * d_new_points;

		hipMallocManaged((void**)& d_points, diagonalSize * sizeof(Point));
		hipMallocManaged((void**)& d_new_points, diagonalSize * sizeof(Point));

		for (int a = 0; a < diagonalSize; a++) {
			d_new_points[a] = Point(-1, -1);
		}

		int* d_data;
		hipMalloc((void**)& d_data, 1 * sizeof(int));
		hipMemset(d_data, 0, 1 * sizeof(int));

		int run_num = 0;

		//Initialize shared memory
		shared_initialize <<<1,1>>>();

		// While there are still points to visit
		while (points[0].getR() != -1 && !pathFound) {

			for (int k = 0; k < diagonalSize; k++) {
				if (points[k].getR() == 0 && points[k].getC() == 0) {
					pathFound = true;
					break;
				} 
				if (points[k].getR() == -1) {
					break;
				}
			}

			hipMemcpy(d_points, &points, diagonalSize * sizeof(Point), hipMemcpyHostToDevice);
			hipMemset(d_data, 0, 1 * sizeof(int));

			ftime(&cuda_start);

			// Call to device function with N threads (at most N points), points to be visited, and array to hold resulting new points to visit for next iteration
			checkPoint << <nb_blocks, nb_threads >> > (d_points, d_new_points, image_copy, nb_threads, d_data, run_num); //d_maze

			hipDeviceSynchronize();

			ftime(&cuda_end);

			cuda_total += 1000 * (cuda_end.time - cuda_start.time) + (cuda_end.millitm - cuda_start.millitm);

			// Copy the resulting new points to visit for next iteration into the points to be visited array
			hipMemcpy(&points, d_new_points, diagonalSize * sizeof(Point), hipMemcpyDeviceToHost);

			run_num++;
			if (run_num % 10 == 0) run_num = 0;
		}

		lodepng_encode32_file(output_filename, image_copy, image_width, image_height);

		// Free cuda copies memory
		hipFree(image_copy);
		hipFree(d_points);
		hipFree(d_new_points);
	}

	// Check that path has been found
	if (pathFound) printf("Path found!\n");
	else printf("Path not found!\n");

	ftime(&end_time);
	total_time = 1000 * (end_time.time - start_time.time) + (end_time.millitm - start_time.millitm);

	printf("Total execution time: %d, Parallel execution time: %d\n", (int)total_time, (int)cuda_total);


	struct timeb s_time, e_time;
	ftime(&s_time);
	postProcessing(output_filename);
	ftime(&e_time);
	float post_time = 1000 * (e_time.time - s_time.time) + (e_time.millitm - s_time.millitm);

	printf("Post processing time: %d", (int)post_time);

	return 0;
}

int findNextIndex(unsigned char* image, int index, int indexValue, int h, int w) {
	int nextIndex = -1;

	if (indexValue == 255) indexValue = 245;

	int r = index / (w * 4);
	int c = index % (w * 4) / 4;

	unsigned current = r * 4 * w + c * 4;
	unsigned right = current + 4;
	unsigned down = current + 4 * w;
	unsigned left = current - 4;
	unsigned up = current - 4 * w;

	if (r == 0 && c == 0) {
		if (image[right] == indexValue + 1) {
			nextIndex = right;
		}
		if (image[down] == indexValue + 1) {
			nextIndex = down;
		}
	}

	else if (r == 0) {
		if (image[right] == indexValue + 1) {
			nextIndex = right;
		}
		if (image[down] == indexValue + 1) {
			nextIndex = down;
		}
		if (image[left] == indexValue + 1) {
			nextIndex = left;
		}
	}

	else if (r == h - 1) {
		if (image[right] == indexValue + 1) {
			nextIndex = right;
		}
		if (image[up] == indexValue + 1) {
			nextIndex = up;
		}
		if (image[left] == indexValue + 1) {
			nextIndex = left;
		}
	}

	else if (c == 0) {
		if (image[up] == indexValue + 1) {
			nextIndex = up;
		}
		if (image[down] == indexValue + 1) {
			nextIndex = down;
		}
		if (image[right] == indexValue + 1) {
			nextIndex = right;
		}
	}

	else if (c == w - 1) {
		if (image[up] == indexValue + 1) {
			nextIndex = up;
		}
		if (image[down] == indexValue + 1) {
			nextIndex = down;
		}
		if (image[left] == indexValue + 1) {
			nextIndex = left;
		}
	}

	else {
		if (image[up] == indexValue + 1) {
			nextIndex = up;
		}
		if (image[down] == indexValue + 1) {
			nextIndex = down;
		}
		if (image[left] == indexValue + 1) {
			nextIndex = left;
		}
		if (image[right] == indexValue + 1) {
			nextIndex = right;
		}
	}

	return nextIndex;
}

void postProcessing(char* inputImage) {
	unsigned error;
	unsigned char* image;
	unsigned image_width, image_height;

	error = lodepng_decode32_file(&image, &image_width, &image_height, inputImage);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));

	int index = 0;

	while (index != image_width * image_height - 1) {
		int indexValue = image[index];
		image[index] = (char)0;
		image[index + 1] = (char)255;

		index = findNextIndex(image, index, indexValue, image_width, image_height);

		if (index == -1) {
			lodepng_encode32_file(inputImage, image, image_width, image_height);
			return;
		}
	}
}